#include "hip/hip_runtime.h"
#include "voxelGrid.h"

/*==================================
Volumetric fusion CUDA
==================================*/
__global__ void fusion_CUDA(voxel *vox, bool *vox_macro_isActive, vec *vertexMap, vec *trans, mat *rot, vec *offset, const int n)
{
	//Compute the index of the voxel
	const int index_vol = n*RESOL_VOL*RESOL_VOL + blockIdx.x * 512 + threadIdx.x;
	const int i = index_vol / (RESOL_VOL*RESOL_VOL);
	const int j = (index_vol % (RESOL_VOL*RESOL_VOL)) / RESOL_VOL;
	const int k = (index_vol % (RESOL_VOL*RESOL_VOL)) % RESOL_VOL;

	//Add offset to the voxel
	int a = i + offset->z;
	int b = j + offset->y;
	int c = k + offset->x;
	const int index = a * RESOL*RESOL + b * RESOL + c;

	//Varify the condition
	if (a < 0 || a > RESOL ||
		b < 0 || b > RESOL ||
		c < 0 || c > RESOL ||
		(vox[index].isActive && vox[index].weight > 8)) return;

	vec tmp, point_vox, dist;

	//Voxel centers in view space
	tmp.x = vox[index].pos.x - trans->x;
	tmp.y = vox[index].pos.y - trans->y;
	tmp.z = vox[index].pos.z - trans->z;

	point_vox.x = rot->x1 * tmp.x + rot->y1 * tmp.y + rot->z1 * tmp.z;
	point_vox.y = rot->x2 * tmp.x + rot->y2 * tmp.y + rot->z2 * tmp.z;
	point_vox.z = rot->x3 * tmp.x + rot->y3 * tmp.y + rot->z3 * tmp.z;

	//Projected onto the img plane
	if (point_vox.z > 0){
		int x = (int)(point_vox.x * FOCAL_LEN / point_vox.z + IMG_WIDTH_OVER_TWO);
		int y = (int)(point_vox.y * -FOCAL_LEN / point_vox.z + IMG_HEIGHT_OVER_TWO);
		int pix = IMG_WIDTH * y + x;

		//The pixel is within the depth map
		if (x > 0 && x < IMG_WIDTH && y > 0 && y < IMG_HEIGHT && vertexMap[pix].z > LOWER_BOUND){
			//Point cloud in view space
			dist.x = vertexMap[pix].x - point_vox.x;
			dist.y = vertexMap[pix].y - point_vox.y;
			dist.z = vertexMap[pix].z - point_vox.z;

			//Compute SDF
			float norm = sqrtf(dist.x * dist.x + dist.y * dist.y + dist.z * dist.z);
			float tsdf = (dist.z > 0) ? norm / TRUNCATE : -norm / TRUNCATE;

			//Truncate SDF
			if (tsdf >= 1)
				vox[index].tsdf = (vox[index].weight * vox[index].tsdf + 1) / (vox[index].weight + 1);
			else if (tsdf > -1){
				vox[index].tsdf = (vox[index].weight * vox[index].tsdf + tsdf) / (vox[index].weight + 1);
				vox[index].isActive = true;
				vox_macro_isActive[(int)((i / VOX_NUM_MACRO) * RESOL_MACRO*RESOL_MACRO) + (int)((j / VOX_NUM_MACRO) * RESOL_MACRO) + (int)(k / VOX_NUM_MACRO)] = true;
			}
			else
				vox[index].tsdf = (vox[index].weight * vox[index].tsdf - 1) / (vox[index].weight + 1);

			++vox[index].weight;
		}
	}
}


/*==================================
Ray-casting for a depth map CUDA
==================================*/
__global__ void rayCast_CUDA(voxel *vox, bool *vox_macro_isActive, vec *vertexMap, vec *vertexMap_fused, vec *bound1, vec *bound2, vec *center, vec *trans, mat *rot)
{
	//Compute the index of the pixel
	const int x = (blockIdx.x * 512 + threadIdx.x) % IMG_WIDTH;
	const int y = (blockIdx.x * 512 + threadIdx.x) / IMG_WIDTH;
	const int pix = IMG_WIDTH * y + x;

	vec tmp, rayDir, rayPos, rayStep, rayStep_macro;
	float norm;
	float tsdf_prev, tsdf_cur = 1;
	bool isActive_prev, isActive_cur = false;

	//Compute ray direction & position
	tmp.x = (x - IMG_WIDTH_OVER_TWO) / FOCAL_LEN;
	tmp.y = (IMG_HEIGHT_OVER_TWO - y) / FOCAL_LEN;
	tmp.z = 1;

	rayDir.x = rot->x1 * tmp.x + rot->x2 * tmp.y + rot->x3 * tmp.z;
	rayDir.y = rot->y1 * tmp.x + rot->y2 * tmp.y + rot->y3 * tmp.z;
	rayDir.z = rot->z1 * tmp.x + rot->z2 * tmp.y + rot->z3 * tmp.z;

	norm = sqrtf(rayDir.x * rayDir.x + rayDir.y * rayDir.y + rayDir.z * rayDir.z);

	//Stepping value per iteration
	rayStep.x = STEP * rayDir.x / norm;
	rayStep.y = STEP * rayDir.y / norm;
	rayStep.z = STEP * rayDir.z / norm;

	rayStep_macro.x = STEP_MACRO * rayDir.x / norm;
	rayStep_macro.y = STEP_MACRO * rayDir.y / norm;
	rayStep_macro.z = STEP_MACRO * rayDir.z / norm;

	rayPos.x = center->x + rayStep.x;
	rayPos.y = center->y + rayStep.y;
	rayPos.z = center->z + rayStep.z;

	//Start casting(macro)
	while (rayPos.x > bound1->x && rayPos.x < bound2->x
	&& rayPos.y > bound1->y && rayPos.y < bound2->y
	&& rayPos.z > bound1->z && rayPos.z < bound2->z){
		//If rayPos is in the volume
		if (rayPos.x > VOX_LEN && rayPos.x < MAX_LEN
		&& rayPos.y > VOX_LEN && rayPos.y < MAX_LEN
		&& rayPos.z > VOX_LEN && rayPos.z < MAX_LEN){
			//If rayPos is in the active zone
			if (vox_macro_isActive[((int)(rayPos.z / VOX_LEN)) / VOX_NUM_MACRO * RESOL_MACRO*RESOL_MACRO + ((int)(rayPos.y / VOX_LEN)) / VOX_NUM_MACRO * RESOL_MACRO + ((int)(rayPos.x / VOX_LEN)) / VOX_NUM_MACRO]){
				rayPos.x -= rayStep_macro.x;
				rayPos.y -= rayStep_macro.y;
				rayPos.z -= rayStep_macro.z;

				//Start casting
				do{
					int index = ((int)(rayPos.z / VOX_LEN)) * RESOL*RESOL + ((int)(rayPos.y / VOX_LEN)) * RESOL + ((int)(rayPos.x / VOX_LEN));

					tsdf_prev = tsdf_cur;
					tsdf_cur = vox[index].tsdf;

					isActive_prev = isActive_cur;
					isActive_cur = vox[index].isActive;

					//Detect a zero-crossing
					if (tsdf_cur * tsdf_prev < 0 && (isActive_cur || isActive_prev)){
						//Interpolation & Transform to the view space
						tmp.x = vox[index].pos.x - vox[index].tsdf * rayDir.x - trans->x;
						tmp.y = vox[index].pos.y - vox[index].tsdf * rayDir.y - trans->y;
						tmp.z = vox[index].pos.z - vox[index].tsdf * rayDir.z - trans->z;

						vertexMap_fused[pix].x = rot->x1 * tmp.x + rot->y1 * tmp.y + rot->z1 * tmp.z;
						vertexMap_fused[pix].y = rot->x2 * tmp.x + rot->y2 * tmp.y + rot->z2 * tmp.z;
						vertexMap_fused[pix].z = rot->x3 * tmp.x + rot->y3 * tmp.y + rot->z3 * tmp.z;
						return;
					}
					rayPos.x += rayStep.x;
					rayPos.y += rayStep.y;
					rayPos.z += rayStep.z;
				} while (rayPos.x > VOX_LEN && rayPos.x < MAX_LEN
				&& rayPos.y > VOX_LEN && rayPos.y < MAX_LEN
				&& rayPos.z > VOX_LEN && rayPos.z < MAX_LEN);
				break;
			}
		}
		rayPos.x += rayStep_macro.x;
		rayPos.y += rayStep_macro.y;
		rayPos.z += rayStep_macro.z;
	}

	//If the ray didn't hit a point, use the raw data
	vertexMap_fused[pix].x = vertexMap[pix].x;
	vertexMap_fused[pix].y = vertexMap[pix].y;
	vertexMap_fused[pix].z = vertexMap[pix].z;
}


/*==================================
Ray-casting (All)
==================================*/
__global__ void rayCastAll_CUDA(voxel *vox, bool *vox_macro_isActive, vec *pc, unsigned int *pc_count)
{
	//Compute the index of the voxel
	const int a = (blockIdx.x * 512 + threadIdx.x) / RESOL;
	const int b = (blockIdx.x * 512 + threadIdx.x) % RESOL;

	float tsdf_prev, tsdf_cur = 0;
	bool isActive_prev, isActive_cur = false;
	int index_tmp, index;

	if(a < RESOL) index_tmp = a * RESOL*RESOL + b * RESOL;
	else if(a < RESOL*2) index_tmp = (a-RESOL) * RESOL*RESOL + b;
	else index = index_tmp = b * RESOL + (a-2*RESOL);

	//Start casting
	for (int c = 1; c < RESOL; ++c){
		if(a < RESOL) index = index_tmp + c;
		else if(a < RESOL*2) index = index_tmp + c * RESOL;
		else index = index_tmp + c * RESOL*RESOL;

		tsdf_prev = tsdf_cur;
		tsdf_cur = vox[index].tsdf;

		isActive_prev = isActive_cur;
		isActive_cur = vox[index].isActive;

		//Detect a zero-crossing
		if ((isActive_cur || isActive_prev) && tsdf_cur * tsdf_prev < 0 && *pc_count < MAX_PC_COUNT){
			//Interpolation
			vec p = vox[index].pos;

			if(a < RESOL) p.z -= vox[index].tsdf;
			else if(a < RESOL*2) p.y -= vox[index].tsdf;
			else p.x -= vox[index].tsdf;

			pc[atomicAdd(pc_count, 1)] = p;
		}
	}
}


/*======================================
Constructor
======================================*/
voxelGrid::voxelGrid()
{
	//Initialize the properties
	center.x = MAX_LEN_VOL / 2.0f;
	center.y = MAX_LEN_VOL / 2.0f;
	center.z = MAX_LEN_VOL / 2.0f;

	dir.x = 0;
	dir.y = 0;
	dir.z = 1;

	trans.x = 0;
	trans.y = 0;
	trans.z = 0;

	rot.x1 = 1; rot.x2 = 0; rot.x3 = 0;
	rot.y1 = 0; rot.y2 = 1; rot.y3 = 0;
	rot.z1 = 0; rot.z2 = 0; rot.z3 = 1;

	//Allocate memory for each voxel
	vox = new voxel[RESOL*RESOL*RESOL];
	vox_macro_isActive = new bool[RESOL_MACRO*RESOL_MACRO*RESOL_MACRO];
	pc = new vec[MAX_PC_COUNT];

	//Initialize each voxel
	for (int i = 0; i < RESOL; i++)
		for (int j = 0; j < RESOL; j++)
			for (int k = 0; k < RESOL; k++){
				int index = i*RESOL*RESOL + j*RESOL + k;

				vox[index].pos.x = k * VOX_LEN + VOX_LEN / 2.f;
				vox[index].pos.y = j * VOX_LEN + VOX_LEN / 2.f,
				vox[index].pos.z = i * VOX_LEN + VOX_LEN / 2.f;
				vox[index].tsdf = 1;
				vox[index].weight = 0;
				vox[index].isActive = false;
			}

	for (int i = 0; i < RESOL_MACRO*RESOL_MACRO*RESOL_MACRO; ++i)
		vox_macro_isActive[i] = false;

	pc_count = 0;

	//Allocate the GPU memory
	hipMalloc(&d_vox, RESOL*RESOL*RESOL * sizeof(voxel));
	hipMalloc(&d_vox_macro_isActive, RESOL_MACRO*RESOL_MACRO*RESOL_MACRO * sizeof(bool));
	hipMalloc(&d_pc, MAX_PC_COUNT * sizeof(vec));
	hipMalloc(&d_vertexMap, IMG_WIDTH*IMG_HEIGHT * sizeof(vec));
	hipMalloc(&d_vertexMap_fused, IMG_WIDTH*IMG_HEIGHT * sizeof(vec));
	hipMalloc(&d_bound1, sizeof(vec));
	hipMalloc(&d_bound2, sizeof(vec));
	hipMalloc(&d_center, sizeof(vec));
	hipMalloc(&d_trans, sizeof(vec));
	hipMalloc(&d_rot, sizeof(mat));
	hipMalloc(&d_offset, sizeof(vec));
	hipMalloc(&d_pc_count, sizeof(unsigned int));

	//Upload data to the GPU memory
	hipMemcpy(d_vox, vox, RESOL*RESOL*RESOL * sizeof(voxel), hipMemcpyHostToDevice);
	hipMemcpy(d_vox_macro_isActive, vox_macro_isActive, RESOL_MACRO*RESOL_MACRO*RESOL_MACRO * sizeof(bool), hipMemcpyHostToDevice);
	hipMemcpy(d_pc, pc, MAX_PC_COUNT * sizeof(vec), hipMemcpyHostToDevice);
	hipMemcpy(d_trans, &trans, sizeof(vec), hipMemcpyHostToDevice);
	hipMemcpy(d_rot, &rot, sizeof(mat), hipMemcpyHostToDevice);
	hipMemcpy(d_offset, &offset, sizeof(vec), hipMemcpyHostToDevice);
	hipMemcpy(d_pc_count, &pc_count, sizeof(unsigned int), hipMemcpyHostToDevice);
}


/*======================================
Destructor
======================================*/
voxelGrid::~voxelGrid()
{
	//Free the GPU memory
	hipFree(d_vox);
	hipFree(d_vox_macro_isActive);
	hipFree(d_pc);
	hipFree(d_vertexMap);
	hipFree(d_vertexMap_fused);
	hipFree(d_bound1);
	hipFree(d_bound2);
	hipFree(d_center);
	hipFree(d_trans);
	hipFree(d_rot);
	hipFree(d_offset);
	hipFree(d_pc_count);

	delete [] vox;
	delete [] vox_macro_isActive;
	delete [] pc;
}


/*======================================
TSDF Fusion
======================================*/
void voxelGrid::fusion(vec *vertexMap)
{
	//Upload data to GPU
	hipMemcpy(d_vertexMap, vertexMap, IMG_WIDTH*IMG_HEIGHT * sizeof(vec), hipMemcpyHostToDevice);

	//Start fusion
	//RESOL_VOL = 128
	//for (int j = 0; j < RESOL_VOL; j += 8)
	//	fusion_CUDA << <512, 512 >> > (d_vox, d_vox_macro_isActive, d_vertexMap, d_trans, d_rot, d_offset, j);

	//RESOL_VOL = 192
	for (int j = 0; j < RESOL_VOL; j += 8)
		fusion_CUDA << <576, 512 >> > (d_vox, d_vox_macro_isActive, d_vertexMap, d_trans, d_rot, d_offset, j);
}


/*======================================
Ray-casting
======================================*/
void voxelGrid::rayCast(vec *vertexMap_fused)
{
	//Compute boundary
	center.x = trans.x;
	center.y = trans.y;
	center.z = trans.z;

	bound1.x = (center.x <= VOX_LEN) ? center.x : VOX_LEN;
	bound1.y = (center.y <= VOX_LEN) ? center.y : VOX_LEN;
	bound1.z = (center.z <= VOX_LEN) ? center.z : VOX_LEN;

	bound2.x = (center.x >= MAX_LEN) ? center.x : MAX_LEN;
	bound2.y = (center.y >= MAX_LEN) ? center.y : MAX_LEN;
	bound2.z = (center.z >= MAX_LEN) ? center.z : MAX_LEN;

	//Upload data to the GPU memory
	hipMemcpy(d_center, &center, sizeof(vec), hipMemcpyHostToDevice);
	hipMemcpy(d_bound1, &bound1, sizeof(vec), hipMemcpyHostToDevice);
	hipMemcpy(d_bound2, &bound2, sizeof(vec), hipMemcpyHostToDevice);

	//Start ray-casting
	rayCast_CUDA << <168, 512 >> > (d_vox, d_vox_macro_isActive, d_vertexMap, d_vertexMap_fused, d_bound1, d_bound2, d_center, d_trans, d_rot);

	//Download data from the GPU memory
	hipMemcpy(vertexMap_fused, d_vertexMap_fused, IMG_WIDTH*IMG_HEIGHT * sizeof(vec), hipMemcpyDeviceToHost);
}


/*======================================
Ray-casting (All: x, y, z direction)
======================================*/
void voxelGrid::rayCastAll(std::vector<vec> &pcData)
{
	//Upload data to GPU
	pc_count = 0;
	hipMemcpy(d_pc_count, &pc_count, sizeof(unsigned int), hipMemcpyHostToDevice);

	rayCastAll_CUDA<<<384, 512>>>(d_vox, d_vox_macro_isActive, d_pc, d_pc_count);

	//Download data from GPU
	hipMemcpy(&pc_count, d_pc_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(pc, d_pc, pc_count * sizeof(vec), hipMemcpyDeviceToHost);

	for(int i = 0; i < pc_count; ++i)
		pcData.push_back(pc[i]);
}


/*======================================
Ray-casting (All: only z direction)
======================================*/
void voxelGrid::rayCastAll_approx(std::vector<vec> &pcData)
{
	//Upload data to GPU
	pc_count = 0;
	hipMemcpy(d_pc_count, &pc_count, sizeof(unsigned int), hipMemcpyHostToDevice);

	rayCastAll_CUDA<<<128, 512>>>(d_vox, d_vox_macro_isActive, d_pc, d_pc_count);

	//Download data from GPU
	hipMemcpy(&pc_count, d_pc_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(pc, d_pc, pc_count * sizeof(vec), hipMemcpyDeviceToHost);

	for(int i = 0; i < pc_count; ++i)
		pcData.push_back(pc[i]);
}


/*======================================
Set translation vector
======================================*/
void voxelGrid::setTrans(float x, float y, float z)
{
	trans.x = x;
	trans.y = y;
	trans.z = z;

	hipMemcpy(d_trans, &trans, sizeof(vec), hipMemcpyHostToDevice);
}


/*======================================
Set rotation matrix
======================================*/
void voxelGrid::setRot(float x1, float x2, float x3,
					   float y1, float y2, float y3,
					   float z1, float z2, float z3)
{
	rot.x1 = x1; rot.x2 = x2; rot.x3 = x3;
	rot.y1 = y1; rot.y2 = y2; rot.y3 = y3;
	rot.z1 = z1; rot.z2 = z2; rot.z3 = z3;

	hipMemcpy(d_rot, &rot, sizeof(mat), hipMemcpyHostToDevice);
}


/*======================================
Compute offset
======================================*/
void voxelGrid::computeOffset()
{
	vec dir_cur;

	//Rotate the orientation of the camera
	dir_cur.x = rot.x1 * dir.x + rot.x2 * dir.y + rot.x3 * dir.z;
	dir_cur.y = rot.y1 * dir.x + rot.y2 * dir.y + rot.y3 * dir.z;
	dir_cur.z = rot.z1 * dir.x + rot.z2 * dir.y + rot.z3 * dir.z;

	//Compute the offset of the voxel index
	offset.x = int((trans.x + SHIFT * dir_cur.x - HALF_LEN_VOL) / VOX_LEN);
	offset.y = int((trans.y + SHIFT * dir_cur.y - HALF_LEN_VOL) / VOX_LEN);
	offset.z = int((trans.z + SHIFT * dir_cur.z - HALF_LEN_VOL) / VOX_LEN);

	//Upload data to the GPU memory
	hipMemcpy(d_offset, &offset, sizeof(vec), hipMemcpyHostToDevice);
}


/*======================================
Get volume offset
======================================*/
vec voxelGrid::getOffsetTrans()
{
	vec offsetTrans;
	offsetTrans.x = offset.x * VOX_LEN;
	offsetTrans.y = offset.y * VOX_LEN;
	offsetTrans.z = offset.z * VOX_LEN;

	return offsetTrans;
}
