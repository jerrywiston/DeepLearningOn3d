#include "hip/hip_runtime.h"
#include "voxelGrid.h"

/*==================================
Volumetric fusion CUDA
==================================*/
__global__ void fusion_CUDA(voxel *vox, vec *vertexMap, vec *trans, mat *rot, vec *offset, const int n)
{
	//Compute the index of the voxel
	const int index_vol = n*RESOL_VOL*RESOL_VOL + blockIdx.x * 512 + threadIdx.x;
	const int i = index_vol / (RESOL_VOL*RESOL_VOL);
	const int j = (index_vol % (RESOL_VOL*RESOL_VOL)) / RESOL_VOL;
	const int k = (index_vol % (RESOL_VOL*RESOL_VOL)) % RESOL_VOL;

	//Add offset to the voxel
	int a = i + offset->z;
	int b = j + offset->y;
	int c = k + offset->x;
	const int index = a * RESOL_Y*RESOL_Z + b * RESOL_Z + c;

	//Varify the condition
	if (a < 0 || a > RESOL_X ||
		b < 0 || b > RESOL_Y ||
		c < 0 || c > RESOL_Z ||
		(vox[index].isActive && vox[index].weight > 8)) return;

	vec tmp, point_vox, dist;

	//Voxel centers in view space
	tmp.x = vox[index].pos.x - trans->x;
	tmp.y = vox[index].pos.y - trans->y;
	tmp.z = vox[index].pos.z - trans->z;

	point_vox.x = rot->x1 * tmp.x + rot->y1 * tmp.y + rot->z1 * tmp.z;
	point_vox.y = rot->x2 * tmp.x + rot->y2 * tmp.y + rot->z2 * tmp.z;
	point_vox.z = rot->x3 * tmp.x + rot->y3 * tmp.y + rot->z3 * tmp.z;

	//Projected onto the img plane
	if (point_vox.z > 0){
		int x = (int)(point_vox.x * FOCAL_LEN / point_vox.z + IMG_WIDTH/2);
		int y = (int)(point_vox.y * -FOCAL_LEN / point_vox.z + IMG_HEIGHT/2);
		int pix = IMG_WIDTH * y + x;

		//The pixel is within the depth map
		if (x > 0 && x < IMG_WIDTH && y > 0 && y < IMG_HEIGHT){
			//Point cloud in view space
			dist.x = vertexMap[pix].x - point_vox.x;
			dist.y = vertexMap[pix].y - point_vox.y;
			dist.z = vertexMap[pix].z - point_vox.z;

			//Compute SDF
			float norm = sqrtf(dist.x * dist.x + dist.y * dist.y + dist.z * dist.z);
			float tsdf = (dist.z > 0) ? norm / TRUNCATE : -norm / TRUNCATE;

			//Truncate SDF
			if (tsdf >= 1)
				vox[index].tsdf = (vox[index].weight * vox[index].tsdf + 1) / (vox[index].weight + 1);
			else if (tsdf > -1){
				vox[index].tsdf = (vox[index].weight * vox[index].tsdf + tsdf) / (vox[index].weight + 1);
				vox[index].isActive = true;
			}
			else
				vox[index].tsdf = (vox[index].weight * vox[index].tsdf - 1) / (vox[index].weight + 1);

			++vox[index].weight;
		}
	}
}


/*==================================
Ray-casting for a depth map CUDA
==================================*/
__global__ void rayCast_CUDA(voxel *vox, vec *vertexMap, vec *vertexMap_fused, vec *bound1, vec *bound2, vec *center, vec *trans, mat *rot)
{
	//Compute the index of the pixel
	const int x = (blockIdx.x * 512 + threadIdx.x) % IMG_WIDTH;
	const int y = (blockIdx.x * 512 + threadIdx.x) / IMG_WIDTH;
	const int pix = IMG_WIDTH * y + x;

	vec tmp, rayDir, rayPos, rayStep;
	float norm;
	float tsdf_prev, tsdf_cur = 1;
	bool isActive_prev, isActive_cur = false;

	//Compute ray direction & position
	tmp.x = (x - IMG_WIDTH/2) / FOCAL_LEN;
	tmp.y = (IMG_HEIGHT/2 - y) / FOCAL_LEN;
	tmp.z = 1;

	rayDir.x = rot->x1 * tmp.x + rot->x2 * tmp.y + rot->x3 * tmp.z;
	rayDir.y = rot->y1 * tmp.x + rot->y2 * tmp.y + rot->y3 * tmp.z;
	rayDir.z = rot->z1 * tmp.x + rot->z2 * tmp.y + rot->z3 * tmp.z;

	norm = sqrtf(rayDir.x * rayDir.x + rayDir.y * rayDir.y + rayDir.z * rayDir.z);

	//Stepping value per iteration
	rayStep.x = STEP * rayDir.x / norm;
	rayStep.y = STEP * rayDir.y / norm;
	rayStep.z = STEP * rayDir.z / norm;

	rayPos.x = center->x + rayStep.x;
	rayPos.y = center->y + rayStep.y;
	rayPos.z = center->z + rayStep.z;

	//Start casting
	//If rayPos is in the volume
	while(rayPos.x > VOX_LEN && rayPos.x < MAX_LEN_X
	   && rayPos.y > VOX_LEN && rayPos.y < MAX_LEN_Y
	   && rayPos.z > VOX_LEN && rayPos.z < MAX_LEN_Z){

		int index = ((int)(rayPos.z / VOX_LEN)) * RESOL_Y*RESOL_Z + ((int)(rayPos.y / VOX_LEN)) * RESOL_Z + ((int)(rayPos.x / VOX_LEN));

		tsdf_prev = tsdf_cur;
		tsdf_cur = vox[index].tsdf;

		isActive_prev = isActive_cur;
		isActive_cur = vox[index].isActive;

		//Detect a zero-crossing
		if (tsdf_cur * tsdf_prev < 0 && (isActive_cur || isActive_prev)){
			//Interpolation & Transform to the view space
			tmp.x = vox[index].pos.x - vox[index].tsdf * rayDir.x - trans->x;
			tmp.y = vox[index].pos.y - vox[index].tsdf * rayDir.y - trans->y;
			tmp.z = vox[index].pos.z - vox[index].tsdf * rayDir.z - trans->z;

			vertexMap_fused[pix].x = rot->x1 * tmp.x + rot->y1 * tmp.y + rot->z1 * tmp.z;
			vertexMap_fused[pix].y = rot->x2 * tmp.x + rot->y2 * tmp.y + rot->z2 * tmp.z;
			vertexMap_fused[pix].z = rot->x3 * tmp.x + rot->y3 * tmp.y + rot->z3 * tmp.z;
			return;
		}
		rayPos.x += rayStep.x;
		rayPos.y += rayStep.y;
		rayPos.z += rayStep.z;
	}

	//If the ray didn't hit a point, use the raw data
	vertexMap_fused[pix].x = vertexMap[pix].x;
	vertexMap_fused[pix].y = vertexMap[pix].y;
	vertexMap_fused[pix].z = vertexMap[pix].z;
}


/*==================================
Ray-casting (All: x direction)
==================================*/
__global__ void rayCastAll_X_CUDA(voxel *vox, vec *pc, unsigned int *pc_count)
{
	//Compute the index of the voxel
	const int a = (blockIdx.x * 512 + threadIdx.x) / RESOL_Z;
	const int b = (blockIdx.x * 512 + threadIdx.x) % RESOL_Z;

	float tsdf_prev, tsdf_cur = 0;
	bool isActive_prev, isActive_cur = false;
	int index_tmp, index;

	index_tmp = a * RESOL_Z + b;

	//Start casting
	for (int c = 1; c < RESOL_X; ++c){
		index = index_tmp + c * RESOL_Y*RESOL_Z;

		tsdf_prev = tsdf_cur;
		tsdf_cur = vox[index].tsdf;

		isActive_prev = isActive_cur;
		isActive_cur = vox[index].isActive;

		//Detect a zero-crossing
		if ((isActive_cur || isActive_prev) && tsdf_cur * tsdf_prev < 0 && *pc_count < MAX_PC_COUNT){
			//Interpolation
			vec p = vox[index].pos;
			p.x -= vox[index].tsdf;

			pc[atomicAdd(pc_count, 1)] = p;
		}
	}
}


/*==================================
Ray-casting (All: y direction)
==================================*/
__global__ void rayCastAll_Y_CUDA(voxel *vox, vec *pc, unsigned int *pc_count)
{
	//Compute the index of the voxel
	const int a = (blockIdx.x * 512 + threadIdx.x) / RESOL_X;
	const int b = (blockIdx.x * 512 + threadIdx.x) % RESOL_X;

	float tsdf_prev, tsdf_cur = 0;
	bool isActive_prev, isActive_cur = false;
	int index_tmp, index;

	index_tmp = a * RESOL_Y*RESOL_Z + b;

	//Start casting
	for (int c = 1; c < RESOL_Y; ++c){
		index = index_tmp + c * RESOL_Z;

		tsdf_prev = tsdf_cur;
		tsdf_cur = vox[index].tsdf;

		isActive_prev = isActive_cur;
		isActive_cur = vox[index].isActive;

		//Detect a zero-crossing
		if ((isActive_cur || isActive_prev) && tsdf_cur * tsdf_prev < 0 && *pc_count < MAX_PC_COUNT){
			//Interpolation
			vec p = vox[index].pos;
			p.y -= vox[index].tsdf;

			pc[atomicAdd(pc_count, 1)] = p;
		}
	}
}


/*==================================
Ray-casting (All: z direction)
==================================*/
__global__ void rayCastAll_Z_CUDA(voxel *vox, vec *pc, unsigned int *pc_count)
{
	//Compute the index of the voxel
	const int a = (blockIdx.x * 512 + threadIdx.x) / RESOL_X;
	const int b = (blockIdx.x * 512 + threadIdx.x) % RESOL_X;

	float tsdf_prev, tsdf_cur = 0;
	bool isActive_prev, isActive_cur = false;
	int index_tmp, index;

	index_tmp = a * RESOL_Y*RESOL_Z + b * RESOL_Z;

	//Start casting
	for (int c = 1; c < RESOL_Z; ++c){
		index = index_tmp + c;

		tsdf_prev = tsdf_cur;
		tsdf_cur = vox[index].tsdf;

		isActive_prev = isActive_cur;
		isActive_cur = vox[index].isActive;

		//Detect a zero-crossing
		if ((isActive_cur || isActive_prev) && tsdf_cur * tsdf_prev < 0 && *pc_count < MAX_PC_COUNT){
			//Interpolation
			vec p = vox[index].pos;
			p.z -= vox[index].tsdf;

			pc[atomicAdd(pc_count, 1)] = p;
		}
	}
}


/*======================================
Constructor
======================================*/
voxelGrid::voxelGrid()
{
	//Initialize the properties
	center.x = MAX_LEN_X / 2.0f;
	center.y = MAX_LEN_Y / 2.0f;
	center.z = MAX_LEN_Z / 2.0f;

	dir.x = 0;
	dir.y = 0;
	dir.z = 1;

	trans.x = 0;
	trans.y = 0;
	trans.z = 0;

	rot.x1 = 1; rot.x2 = 0; rot.x3 = 0;
	rot.y1 = 0; rot.y2 = 1; rot.y3 = 0;
	rot.z1 = 0; rot.z2 = 0; rot.z3 = 1;

	//Allocate memory for each voxel
	vox = new voxel[RESOL_X*RESOL_Y*RESOL_Z];
	pc = new vec[MAX_PC_COUNT];

	//Initialize each voxel
	for (int i = 0; i < RESOL_X; i++)
		for (int j = 0; j < RESOL_Y; j++)
			for (int k = 0; k < RESOL_Z; k++){
				int index = i*RESOL_Y*RESOL_Z + j*RESOL_Z + k;

				vox[index].pos.x = k * VOX_LEN + VOX_LEN / 2.f;
				vox[index].pos.y = j * VOX_LEN + VOX_LEN / 2.f,
				vox[index].pos.z = i * VOX_LEN + VOX_LEN / 2.f;
				vox[index].tsdf = 1;
				vox[index].weight = 0;
				vox[index].isActive = false;
			}

	pc_count = 0;

	//Allocate the GPU memory
	hipMalloc(&d_vox, RESOL_X*RESOL_Y*RESOL_Z * sizeof(voxel));
	hipMalloc(&d_pc, MAX_PC_COUNT * sizeof(vec));
	hipMalloc(&d_vertexMap, IMG_WIDTH*IMG_HEIGHT * sizeof(vec));
	hipMalloc(&d_vertexMap_fused, IMG_WIDTH*IMG_HEIGHT * sizeof(vec));
	hipMalloc(&d_bound1, sizeof(vec));
	hipMalloc(&d_bound2, sizeof(vec));
	hipMalloc(&d_center, sizeof(vec));
	hipMalloc(&d_trans, sizeof(vec));
	hipMalloc(&d_rot, sizeof(mat));
	hipMalloc(&d_offset, sizeof(vec));
	hipMalloc(&d_pc_count, sizeof(unsigned int));

	//Upload data to the GPU memory
	hipMemcpy(d_vox, vox, RESOL_X*RESOL_Y*RESOL_Z * sizeof(voxel), hipMemcpyHostToDevice);
	hipMemcpy(d_pc, pc, MAX_PC_COUNT * sizeof(vec), hipMemcpyHostToDevice);
	hipMemcpy(d_trans, &trans, sizeof(vec), hipMemcpyHostToDevice);
	hipMemcpy(d_rot, &rot, sizeof(mat), hipMemcpyHostToDevice);
	hipMemcpy(d_offset, &offset, sizeof(vec), hipMemcpyHostToDevice);
	hipMemcpy(d_pc_count, &pc_count, sizeof(unsigned int), hipMemcpyHostToDevice);
}


/*======================================
Destructor
======================================*/
voxelGrid::~voxelGrid()
{
	//Free the GPU memory
	hipFree(d_vox);
	hipFree(d_pc);
	hipFree(d_vertexMap);
	hipFree(d_vertexMap_fused);
	hipFree(d_bound1);
	hipFree(d_bound2);
	hipFree(d_center);
	hipFree(d_trans);
	hipFree(d_rot);
	hipFree(d_offset);
	hipFree(d_pc_count);

	delete [] vox;
	delete [] pc;
}


/*======================================
TSDF Fusion
======================================*/
void voxelGrid::fusion(vec *vertexMap)
{
	//Upload data to GPU
	hipMemcpy(d_vertexMap, vertexMap, IMG_WIDTH*IMG_HEIGHT * sizeof(vec), hipMemcpyHostToDevice);

	//Start fusion
	//RESOL_VOL = 128
	//for (int j = 0; j < RESOL_VOL; j += 8)
	//	fusion_CUDA << <512, 512 >> > (d_vox, d_vertexMap, d_trans, d_rot, d_offset, j);

	//RESOL_VOL = 192
	//for (int j = 0; j < RESOL_VOL; j += 8)
	//	fusion_CUDA << <576, 512 >> > (d_vox, d_vertexMap, d_trans, d_rot, d_offset, j);

	//RESOL_VOL = 256
	for (int j = 0; j < RESOL_VOL; j += 4)
		fusion_CUDA << <512, 512 >> > (d_vox, d_vertexMap, d_trans, d_rot, d_offset, j);
}


/*======================================
Ray-casting
======================================*/
void voxelGrid::rayCast(vec *vertexMap_fused)
{
	//Compute boundary
	center.x = trans.x;
	center.y = trans.y;
	center.z = trans.z;

	bound1.x = (center.x <= VOX_LEN) ? center.x : VOX_LEN;
	bound1.y = (center.y <= VOX_LEN) ? center.y : VOX_LEN;
	bound1.z = (center.z <= VOX_LEN) ? center.z : VOX_LEN;

	bound2.x = (center.x >= MAX_LEN_X) ? center.x : MAX_LEN_X;
	bound2.y = (center.y >= MAX_LEN_Y) ? center.y : MAX_LEN_Y;
	bound2.z = (center.z >= MAX_LEN_Z) ? center.z : MAX_LEN_Z;

	//Upload data to the GPU memory
	hipMemcpy(d_center, &center, sizeof(vec), hipMemcpyHostToDevice);
	hipMemcpy(d_bound1, &bound1, sizeof(vec), hipMemcpyHostToDevice);
	hipMemcpy(d_bound2, &bound2, sizeof(vec), hipMemcpyHostToDevice);

	//Start ray-casting
	rayCast_CUDA << <168, 512 >> > (d_vox, d_vertexMap, d_vertexMap_fused, d_bound1, d_bound2, d_center, d_trans, d_rot);

	//Download data from the GPU memory
	hipMemcpy(vertexMap_fused, d_vertexMap_fused, IMG_WIDTH*IMG_HEIGHT * sizeof(vec), hipMemcpyDeviceToHost);
}


/*======================================
Ray-casting (All: x, y, z direction)
======================================*/
void voxelGrid::rayCastAll(std::vector<vec> &pcData)
{
	//Upload data to GPU
	pc_count = 0;
	hipMemcpy(d_pc_count, &pc_count, sizeof(unsigned int), hipMemcpyHostToDevice);

	rayCastAll_X_CUDA<<<256, 512>>>(d_vox, d_pc, d_pc_count);
	rayCastAll_Y_CUDA<<<512, 512>>>(d_vox, d_pc, d_pc_count);
	rayCastAll_Z_CUDA<<<256, 512>>>(d_vox, d_pc, d_pc_count);

	//Download data from GPU
	hipMemcpy(&pc_count, d_pc_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(pc, d_pc, pc_count * sizeof(vec), hipMemcpyDeviceToHost);

	for(int i = 0; i < pc_count; ++i)
		pcData.push_back(pc[i]);
}


/*======================================
Ray-casting (All: only z direction)
======================================*/
void voxelGrid::rayCastAll_approx(std::vector<vec> &pcData)
{
	//Upload data to GPU
	pc_count = 0;
	hipMemcpy(d_pc_count, &pc_count, sizeof(unsigned int), hipMemcpyHostToDevice);

	rayCastAll_X_CUDA<<<208, 512>>>(d_vox, d_pc, d_pc_count);

	//Download data from GPU
	hipMemcpy(&pc_count, d_pc_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(pc, d_pc, pc_count * sizeof(vec), hipMemcpyDeviceToHost);

	//pcData.insert(pcData.being(), pc, pc+pc_count);

	for(int i = 0; i < pc_count; ++i)
		pcData.push_back(pc[i]);
}


/*======================================
Set translation vector
======================================*/
void voxelGrid::setTrans(float x, float y, float z)
{
	trans.x = x;
	trans.y = y;
	trans.z = z;

	hipMemcpy(d_trans, &trans, sizeof(vec), hipMemcpyHostToDevice);
}

void voxelGrid::setTrans(vec t)
{
	trans = t;
	hipMemcpy(d_trans, &trans, sizeof(vec), hipMemcpyHostToDevice);
}


/*======================================
Set rotation matrix
======================================*/
void voxelGrid::setRot(float x1, float x2, float x3,
					   float y1, float y2, float y3,
					   float z1, float z2, float z3)
{
	rot.x1 = x1; rot.x2 = x2; rot.x3 = x3;
	rot.y1 = y1; rot.y2 = y2; rot.y3 = y3;
	rot.z1 = z1; rot.z2 = z2; rot.z3 = z3;

	hipMemcpy(d_rot, &rot, sizeof(mat), hipMemcpyHostToDevice);
}

void voxelGrid::setRot(mat r)
{
	rot = r;
	hipMemcpy(d_rot, &rot, sizeof(mat), hipMemcpyHostToDevice);
}


/*======================================
Compute offset
======================================*/
void voxelGrid::computeOffset()
{
	vec dir_cur;

	//Rotate the orientation of the camera
	dir_cur.x = rot.x1 * dir.x + rot.x2 * dir.y + rot.x3 * dir.z;
	dir_cur.y = rot.y1 * dir.x + rot.y2 * dir.y + rot.y3 * dir.z;
	dir_cur.z = rot.z1 * dir.x + rot.z2 * dir.y + rot.z3 * dir.z;

	//Compute the offset of the voxel index
	offset.x = int((trans.x + SHIFT * dir_cur.x - MAX_LEN_VOL/2) / VOX_LEN);
	offset.y = int((trans.y + SHIFT * dir_cur.y - MAX_LEN_VOL/2) / VOX_LEN);
	offset.z = int((trans.z + SHIFT * dir_cur.z - MAX_LEN_VOL/2) / VOX_LEN);

	//Upload data to the GPU memory
	hipMemcpy(d_offset, &offset, sizeof(vec), hipMemcpyHostToDevice);
}


/*======================================
Get volume offset
======================================*/
vec voxelGrid::getOffsetTrans()
{
	vec offsetTrans;
	offsetTrans.x = offset.x * VOX_LEN;
	offsetTrans.y = offset.y * VOX_LEN;
	offsetTrans.z = offset.z * VOX_LEN;

	return offsetTrans;
}


/*====================================
Compute TSDF from the point cloud
====================================*/
void voxelGrid::inverseTSDF(std::vector<vec> &pc)
{
	vec vertexMap[IMG_WIDTH*IMG_HEIGHT];

	for(int k = 0; k < 16; ++k)
	for(int j = 0; j < 16; ++j){
		for(int i = 0; i < IMG_WIDTH*IMG_HEIGHT; ++i){
			vertexMap[i].x = 0;
			vertexMap[i].y = 0;
			vertexMap[i].z = 0;
		}

		setTrans(256*j, 256*k, 2048);
		setRot(1, 0, 0,
			   0, 1, 0,
		       0, 0, 1);

		for(int i = 0; i < pc.size(); ++i){
			vec tmp, point, point_viewSpace;

			point = pc[i];

			//Varify the condition
			if(point.x < 0 || point.x > MAX_LEN_X ||
			   point.y < 0 || point.y > MAX_LEN_Y ||
		   	   point.z < 0 || point.z > MAX_LEN_Z) continue;

			//Point in view space
			tmp.x = point.x - trans.x;
			tmp.y = point.y - trans.y;
			tmp.z = point.z - trans.z;

			point_viewSpace.x = rot.x1 * tmp.x + rot.y1 * tmp.y + rot.z1 * tmp.z;
			point_viewSpace.y = rot.x2 * tmp.x + rot.y2 * tmp.y + rot.z2 * tmp.z;
			point_viewSpace.z = rot.x3 * tmp.x + rot.y3 * tmp.y + rot.z3 * tmp.z;

			//Projected onto the img plane
			if(point_viewSpace.z > 0){
				int x = (int)(point_viewSpace.x * FOCAL_LEN / point_viewSpace.z + IMG_WIDTH/2);
				int y = (int)(point_viewSpace.y * -FOCAL_LEN / point_viewSpace.z + IMG_HEIGHT/2);
				int pix = IMG_WIDTH * y + x;

				//The pixel is within the depth map
				if (x > 0 && x < IMG_WIDTH && y > 0 && y < IMG_HEIGHT){
					vertexMap[pix] = point_viewSpace;
				}
			}
		}

		computeOffset();
		fusion(vertexMap);
	}
}


voxel* voxelGrid::getVox()
{
	//Download voxel data
	hipMemcpy(vox, d_vox, RESOL_X*RESOL_Y*RESOL_Z * sizeof(voxel), hipMemcpyDeviceToHost);

	return vox;
}
